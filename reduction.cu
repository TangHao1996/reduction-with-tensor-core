#include "hip/hip_runtime.h"
#include <assert.h>
#include <stdio.h>
#include <sstream>
#include <iostream>
#include<time.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <mma.h>
#include <hip/hip_runtime_api.h>
#include <hipcub/hipcub.hpp> //the CUDA unbound library unbrella head file

#define WMMA_M 16
#define WMMA_N 16
#define WMMA_K 16


#define SHMEM_SIZE 32*1024
#define WARP_SIZE 32
#define WARPS_PER_BLOCK 16
#define THREADS_PER_BLOCK (WARP_SIZE * WARPS_PER_BLOCK)

#define CONST_BYTES (16*16*2)
#define SKEW_HALF 8                      //offset for avoding bank conflict
#define SHMEM_STRIDE (16+SKEW_HALF)
#define INPUT_STORE_POINT WMMA_M
#define FREE_USE (INPUT_STORE_POINT+16)

#define checkCudaErrors(status) {                                      \
    std::stringstream _error;                                          \
    if (status != 0) {                                                 \
      _error << "Cuda failure\nError: " << hipGetErrorString(status); \
      std::stringstream _where, _message;                                \
      _where << __FILE__ << ':' << __LINE__;                             \
      _message << _error.str() + "\n" << __FILE__ << ':' << __LINE__;\
      std::cerr << _message.str() << "\nAborting...\n";                  \
      hipDeviceReset();                                                 \
      exit(EXIT_FAILURE);                                                \
    }                                                                  \
}

#define checkKernelErrors(expr)                             \
  do {                                                      \
    expr;                                                   \
                                                            \
    hipError_t __err = hipGetLastError();                 \
    if (__err != hipSuccess) {                             \
      printf("Line %d: '%s' failed: %s\n", __LINE__, #expr, \
             hipGetErrorString(__err));                    \
      abort();                                              \
    }                                                       \
  } while (0)



using namespace nvcuda;
#define frag_c wmma::fragment<wmma::accumulator, 16, 16, 16, half>
//using frag_b = wmma::fragment<wmma::matrix_b, 16, 16, 16, half, wmma::row_major>;
// malloc gpu constant memory
__constant__  half P_d[16*16];

half P_h[16*16]={1.0,1.0,1.0,1.0,1.0,1.0,1.0,1.0,1.0,1.0,1.0,1.0,1.0,1.0,1.0,1.0,
                0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,
                0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,
                0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,
                0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,
                0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,
                0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,
                0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,
                0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,
                0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,
                0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,
                0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,
                0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,
                0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,
                0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,
                0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0
                };


__global__ void compute_reductions16N_warp(const half *input, float *output, int N){

    extern __shared__ half shmem[][16 + SKEW_HALF];
    half *free_use = (half*)&shmem[FREE_USE][0];

    const unsigned int warpId = threadIdx.x / WARP_SIZE;
    const unsigned int laneId = threadIdx.x % WARP_SIZE;

    if(warpId==0){

      typedef int4 copy_t;//vector pointer for fast copy
      //load P matrix to shared memory
      int shmem_row = laneId/2;
      //just remember this fixed copy approach!!
      copy_t *lane_ptr = (copy_t *)(P_d+laneId*8);          //one thread copy a int4 = 16bytes = 8 fp16.
      *((copy_t *)&shmem[shmem_row][0]+laneId%2) = *lane_ptr;
      
      //load input
      if(laneId < N<<1){
        lane_ptr = (copy_t *)(input+laneId*8);
        *((copy_t *)&shmem[INPUT_STORE_POINT+shmem_row][0]+laneId%2) = *lane_ptr;
      }
      else{
        *((copy_t *)&shmem[INPUT_STORE_POINT+shmem_row][0]+laneId%2) = make_int4(0,0,0,0);//padding with 0;
      }

       __syncthreads();  


      wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, half, wmma::row_major> P_frag;
      wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, half, wmma::col_major> PT_frag;
      wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, half, wmma::col_major> A_frag;
      wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, half, wmma::row_major> V_frag;
      wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, float> C_frag;
      wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, half> Vn_frag;
      wmma::fill_fragment(C_frag, 0.0f);

      
      wmma::load_matrix_sync(P_frag, &shmem[0][0], SHMEM_STRIDE);
      wmma::load_matrix_sync(PT_frag, &shmem[0][0], SHMEM_STRIDE);
      //wmma::load_matrix_sync(A_frag, &shmem[INPUT_STORE_POINT][0], SHMEM_STRIDE);
      wmma::load_matrix_sync(A_frag, &shmem[INPUT_STORE_POINT][0], SHMEM_STRIDE);

      wmma::mma_sync(Vn_frag, P_frag, A_frag, C_frag);//perform V = P x A

      wmma::store_matrix_sync(free_use, Vn_frag, 16, wmma::mem_row_major);//store Vn to shared memory
      wmma::load_matrix_sync(V_frag, free_use, 16);//load V from Vn

      wmma::mma_sync(C_frag, V_frag, PT_frag, C_frag);//perform output = V x PT 
      
      wmma::store_matrix_sync(output, C_frag, 16, wmma::mem_row_major);
    }

    //if(threadIdx.x==0)
      //printf("kernel complete!\n");
}

/************************
 * WARP-LEVEL REDUCTION *
 ************************/
__global__ void compute_reductions256N_warp(const half *input, float *output, int N){

  extern __shared__ half shmem[][16 + SKEW_HALF];
  half *free_use = (half*)&shmem[FREE_USE][0];
  const unsigned int warpId = threadIdx.x / WARP_SIZE;
  //const unsigned int laneId = threadIdx.x % WARP_SIZE;


  if(warpId==0){

     __syncthreads();  

    wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, half, wmma::row_major> P_frag;
    wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, half, wmma::col_major> PT_frag;
    wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, half, wmma::col_major> A_frag;
    wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, half, wmma::row_major> V_frag;
    wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, float> C_frag;
    wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, half> Vn_frag;
    wmma::fill_fragment(C_frag, 0.0f);
    wmma::fill_fragment(Vn_frag, 0.0f);
    wmma::fill_fragment(P_frag, 1.0f);
    wmma::fill_fragment(PT_frag, 1.0f);


#pragma unroll
    for(int i=0;i<N;i++){
      wmma::load_matrix_sync(A_frag, input+i*256, 16);  
      wmma::mma_sync(Vn_frag, P_frag, A_frag, Vn_frag);                 //perform Vn = P x An+Vn-1
    }

    wmma::store_matrix_sync(free_use, Vn_frag, 16, wmma::mem_row_major);//store Vn to shared memory, because as an accumulator frag Vn cannot be used for computing multiplication
    wmma::load_matrix_sync(V_frag, free_use, 16);                       //load V from Vn as a matrix_a type   
    wmma::mma_sync(C_frag, V_frag, PT_frag, C_frag);                    //perform output = V x PT 
    
    if(threadIdx.x == 0)
      *output = C_frag.x[0];
  }
}


/*************************
 * BLOCK-LEVEL REDUCTION *
 *************************/
__global__ void compute_reductions256N_block(const half *input, float *output, int N){
  extern __shared__ half shmem[][16 + SKEW_HALF];
  const unsigned int warpId = threadIdx.x / WARP_SIZE;
  const unsigned int laneId = threadIdx.x % WARP_SIZE;
  half *free_use = (half*)&shmem[FREE_USE+warpId*16][0];//shared memory buffer start point for each warp
  float *partial_sums = (float *)&shmem[0][0];
  
  if(warpId==0){
    typedef int4 copy_t;                                  //vector pointer for fast copy
    //load P matrix to shared memory
    int shmem_row = laneId/2;
    //just remember this fixed copy approach!!
    copy_t *lane_ptr = (copy_t *)(P_d+laneId*8);          //one thread copy a int4 = 16bytes = 8 fp16
    *((copy_t *)&shmem[shmem_row][0]+laneId%2) = *lane_ptr;  
  }
  __syncthreads();                                        //如果这里不加, 其他warp就会先执行下面的语句以至于读不到shared memory里的数据
  
  wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, half, wmma::row_major> P_frag;
  wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, half, wmma::col_major> PT_frag;
  wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, half, wmma::col_major> A_frag;
  wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, half, wmma::row_major> V_frag;
  wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, float> C_frag;
  wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, half> Vn_frag;
  wmma::fill_fragment(C_frag, 0.0f);
  wmma::fill_fragment(Vn_frag, 0.0f);
  wmma::load_matrix_sync(P_frag, &shmem[0][0], SHMEM_STRIDE);
  wmma::load_matrix_sync(PT_frag, &shmem[0][0], SHMEM_STRIDE);

  unsigned int i=0;
  while(warpId+i < N){
    wmma::load_matrix_sync(A_frag, input+(warpId+i)*256, 16);
    wmma::mma_sync(Vn_frag, P_frag, A_frag, Vn_frag);                 //perform Vn = P x An+Vn-1
    i+=WARPS_PER_BLOCK;
  } 

  wmma::store_matrix_sync(free_use, Vn_frag, 16, wmma::mem_row_major);//store Vn to shared memory, because as an accumulator frag Vn cannot be used for computing multiplication
  wmma::load_matrix_sync(V_frag, free_use, 16);                       //load V from Vn as a matrix_a type
  wmma::mma_sync(C_frag, V_frag, PT_frag, C_frag);                    //perform output = V x PT
  __syncthreads();                                                    

  
  if(laneId == 0){
    partial_sums[warpId] = C_frag.x[0];
  }     
  __syncthreads();
  
  if(warpId == 0){
    float mysum = 0.0f;
    if(threadIdx.x < 16){
      mysum = partial_sums[threadIdx.x];
#pragma unroll 
      for(int offset = 8; offset > 0; offset >>= 1)
        mysum += __shfl_down_sync(0xffffffff, mysum, offset, 8);
      //printf("%f, ", mysum);
    }
    if(threadIdx.x == 0)
      *output = mysum;
  }

}

/*******************************************
 * BLOCK LEVEL REDCUTION WITH OPTIMIZATION *
 *******************************************/
__global__ void compute_reductions256N_block_opt(const half *input, float *output, int N){

  __shared__ half res_warps[16*256];
  __shared__ float partial_sums[256];

  const unsigned int warpId = threadIdx.x / WARP_SIZE;
  //const unsigned int laneId = threadIdx.x % WARP_SIZE;
  half *res_ptr = &(res_warps[warpId*256]);

  
  wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, half, wmma::row_major> P_frag;
  wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, half, wmma::col_major> PT_frag;
  wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, half, wmma::col_major> A_frag;
  wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, half, wmma::row_major> V_frag;
  wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, float> C_frag;
  wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, half> Vn_frag;
  wmma::fill_fragment(C_frag, 0.0f);
  wmma::fill_fragment(Vn_frag, 0.0f);
  wmma::fill_fragment(PT_frag, 1.0f);
  wmma::fill_fragment(P_frag, 1.0f);


  unsigned int i=0;
#pragma unroll
  while(warpId+i < N){
    wmma::load_matrix_sync(A_frag, input+(warpId+i)*256, 16);
    wmma::mma_sync(Vn_frag, A_frag, PT_frag, Vn_frag);
    i += WARPS_PER_BLOCK;
  }

  wmma::store_matrix_sync(res_ptr, Vn_frag, 16, wmma::mem_col_major);                //store Vn to shared memory, because as an accumulator frag Vn cannot be used for computing multiplication
  __syncthreads();
  if(warpId == 0){
    wmma::load_matrix_sync(V_frag, res_ptr, 256);
    /*
    for(int i = 0;i<V_frag.num_elements;i++)
      printf("%.2f, ", __half2float(V_frag.x[i]));
    /*
    if(threadIdx.x == 0)
      for(int i = 0;i<256;i++)
        printf("%.2f, ", __half2float(res_ptr[256+i]));
    */
    wmma::mma_sync(C_frag, P_frag, V_frag, C_frag);
    wmma::store_matrix_sync(partial_sums, C_frag, 16, wmma::mem_row_major);

  //__syncthreads();
    float mysum = 0.0f;
    if(threadIdx.x < 16)
      mysum = partial_sums[threadIdx.x];
#pragma unroll 
    for(int offset = 8; offset > 0; offset >>= 1)
      mysum += __shfl_down_sync(0xffffffff, mysum, offset, 16);
      //printf("%f, ", mysum);

    if(threadIdx.x == 0)
      *output = mysum;
  }

}

/******************************************
 * BLOCK LEVEL REDUCTION WITH CUB LIBRARY *
 ******************************************/
 template <
    int                     BLOCK_THREADS,
    int                     ITEMS_PER_THREAD,
    hipcub::BlockReduceAlgorithm    ALGORITHM,
    class T>
__global__ void BlockSumKernel(
    T             *d_in,          // Tile of input
    float         *d_out)         // Tile aggregate
{
    // Specialize BlockReduce type for our thread block
    typedef hipcub::BlockReduce<T, BLOCK_THREADS, ALGORITHM> BlockReduceT;
    // Shared memory
    __shared__ typename BlockReduceT::TempStorage temp_storage;
    // Per-thread tile data
    T data[ITEMS_PER_THREAD];
    hipcub::LoadDirectStriped<BLOCK_THREADS>(threadIdx.x, d_in, data);
    // Compute sum
    float aggregate = BlockReduceT(temp_storage).Sum(data);
    // Store aggregate
    if (threadIdx.x == 0)
    {
        *d_out = aggregate;
    }
}


/*******************************
 * GRID LEVEL REDUCTION KERNEL *
 *******************************/
template<int BLOCKS_PER_GIRD>
__global__ void compute_reductions256N_grid(half *input, float *output, int N){


  const unsigned int blockId = blockIdx.x;
  const unsigned int warpId = threadIdx.x / WARP_SIZE;

  __shared__ half res_warps[16*256];
  __shared__ float partial_sums[256];
  half *res_ptr = &(res_warps[warpId*256]);

  wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, half, wmma::row_major> P_frag;
  wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, half, wmma::col_major> PT_frag;
  wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, half, wmma::col_major> A_frag;
  wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, half, wmma::row_major> V_frag;
  wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, float> C_frag;
  wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, half> Vn_frag;
  wmma::fill_fragment(C_frag, 0.0f);
  wmma::fill_fragment(Vn_frag, 0.0f);
  wmma::fill_fragment(PT_frag, 1.0f);
  wmma::fill_fragment(P_frag, 1.0f);


  unsigned int i=WARPS_PER_BLOCK * blockId+warpId;
  
#pragma unroll
  while(i < N){
    wmma::load_matrix_sync(A_frag, input+i*256, 16);
    wmma::mma_sync(Vn_frag, A_frag, PT_frag, Vn_frag);
    i += (WARPS_PER_BLOCK * BLOCKS_PER_GIRD);
  }

  wmma::store_matrix_sync(res_ptr, Vn_frag, 16, wmma::mem_col_major);                //store Vn to shared memory, because as an accumulator frag Vn cannot be used for computing multiplication  
  __syncthreads();

  if(warpId == 0){
    wmma::load_matrix_sync(V_frag, res_ptr, 256);
    wmma::mma_sync(C_frag, P_frag, V_frag, C_frag);
    wmma::store_matrix_sync(partial_sums, C_frag, 16, wmma::mem_row_major);

  //__syncthreads();
    float mysum = 0.0f;
    if(threadIdx.x < 16)
      mysum = partial_sums[threadIdx.x];
#pragma unroll 
    for(int offset = 8; offset > 0; offset >>= 1)
      mysum += __shfl_down_sync(0xffffffff, mysum, offset, 16);
      //printf("%f, ", mysum);

    if(threadIdx.x == 0)
      output[blockId] = mysum;
  }

}



/**********************************
 * TEST MEMORY TRANSFER SPEED *
 **********************************/
__global__ void mem_test(const half *input){
  extern __shared__ half shmem[][16 + SKEW_HALF];
  const unsigned int warpId = threadIdx.x / WARP_SIZE;
  //const unsigned int laneId = threadIdx.x % WARP_SIZE;

  if(warpId==0){

    typedef int4 copy_t;                                              //vector pointer for fast copy
    //load P matrix to shared memory
    //int shmem_row = laneId/2;
    //copy_t *lane_ptr = (copy_t *)(P_d+laneId*8);
    //copy_t *lane_ptr = (copy_t *)(input+laneId*8);                  //one thread copy a int4 = 16bytes = 8 fp16

    //constant memory ----> shared memory      
    //*((copy_t *)&shmem[shmem_row][0]+laneId%2) = *lane_ptr;

    //global memory ----> shared memory
    //*((copy_t *)&shmem[shmem_row][0]+laneId%2) = *lane_ptr;
    
    
  }  
  //directly set the shared memory
  //if(threadIdx.x < 16)
    //*(&shmem[0][0] + threadIdx.x) = __float2half(1.0f);
  //else
    //*(&shmem[0][0] + threadIdx.x) = __float2half(0.0f);

  //__syncthreads();
  wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, half, wmma::row_major> P_frag;
  //load from shared memory
  //wmma::load_matrix_sync(P_frag, &shmem[0][0], SHMEM_STRIDE);
  //directly load from global memory
  //wmma::load_matrix_sync(P_frag, input, SHMEM_STRIDE);
  //directly fill the frag
  wmma::fill_fragment(P_frag, 1.0f);

}




__host__ void init_input(half *input_half, float *input_float,int size){
  srand((int)time(0));
  for(int i=0;i<size;i++){
    input_float[i] = (float)(rand() % 100);
    //input_half[i] = __float2half(((float)(input_float[i])-1.0f+0.25f));
    input_half[i] = __float2half(((float)(input_float[i]))/100000.0f);
  }
}


 template<int BLOCKS_PER_GIRD>
 float sum_device(const half *input, const int input_size){
 
   half *input_d;
   float *res_seq;
   float *res_d;
   float res_h = 0;

   checkCudaErrors(hipMalloc(&input_d, 2*input_size));
   checkCudaErrors(hipMalloc(&res_d, sizeof(float)));
   checkCudaErrors(hipMalloc(&res_seq, BLOCKS_PER_GIRD*sizeof(float)));
 
   checkCudaErrors(hipMemcpy(input_d, input, 2*input_size, hipMemcpyHostToDevice));
 
   checkKernelErrors( (compute_reductions256N_grid<BLOCKS_PER_GIRD><<<BLOCKS_PER_GIRD, THREADS_PER_BLOCK>>>(input_d, res_seq, input_size/256)) );
   checkCudaErrors(hipDeviceSynchronize());
   BlockSumKernel<BLOCKS_PER_GIRD, 1, hipcub::BLOCK_REDUCE_RAKING, float><<<1, BLOCKS_PER_GIRD>>>(res_seq, res_d);
  
   checkCudaErrors(hipMemcpy(&res_h, res_d, sizeof(float), hipMemcpyDeviceToHost));

   checkCudaErrors(hipFree(input_d));
   checkCudaErrors(hipFree(res_d));
   checkCudaErrors(hipFree(res_seq));

   return res_h;
 }


int sum_wmma(half *input, int input_size){
  float res_h = 0.0;
  float *res_d;

  half *input_d;
  //malloc GPU and copy contant data to constant memory
  checkCudaErrors(hipMalloc(&input_d, 2*input_size));
  checkCudaErrors(hipMalloc(&res_d, sizeof(float)));
  checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(P_d), P_h, CONST_BYTES));

  //copy input to gpu
  checkCudaErrors(hipMemcpy(input_d, input, 2*input_size, hipMemcpyHostToDevice));

  //launch kernel
  checkKernelErrors( (compute_reductions256N_block<<<1, THREADS_PER_BLOCK, SHMEM_SIZE>>>(input_d, res_d, input_size/256)) );
  //checkKernelErrors( (compute_reductions256N_block_opt<<<1, THREADS_PER_BLOCK>>>(input_d, res_d, input_size/256)) );
  //checkKernelErrors( (compute_reductions256N_warp<<<1, 32, SHMEM_SIZE>>>(input_d, res_d, input_size/256)) );

  //checkCudaErrors(hipDeviceSynchronize());
  //checkKernelErrors( (mem_test<<<1, THREADS_PER_BLOCK, SHMEM_SIZE>>>(input_d)) );

  //copy result back to cpu
  checkCudaErrors(hipMemcpy(&res_h, res_d, sizeof(float), hipMemcpyDeviceToHost));

  //free gpu memory and return
  checkCudaErrors(hipFree(input_d));
  checkCudaErrors(hipFree(res_d));
  return res_h;
}

template<class T>
int sum_cub(T *input, int input_size){
  float res_h = 0;
  float *res_d;
  T *input_d;
  checkCudaErrors(hipMalloc(&input_d, sizeof(T)*input_size));
  checkCudaErrors(hipMalloc(&res_d, sizeof(float)));
  checkCudaErrors(hipMemcpy(input_d, input, sizeof(T)*input_size, hipMemcpyHostToDevice));
  BlockSumKernel<THREADS_PER_BLOCK, 1<<2, hipcub::BLOCK_REDUCE_RAKING, T><<<1, THREADS_PER_BLOCK>>>(input_d, res_d);
  checkCudaErrors(hipMemcpy(&res_h, res_d, sizeof(float), hipMemcpyDeviceToHost));
  checkCudaErrors(hipFree(input_d));
  checkCudaErrors(hipFree(res_d));
  return res_h;
}

int main(){

    int N = 1<<16;
    int input_size = N*256;

    half *input_h = (half*)malloc(2*input_size);
    float *input_h_cub = (float*)malloc(4*input_size);
    init_input(input_h, input_h_cub, input_size);

    //float res_cub = sum_cub<float>(input_h_cub, input_size);
    //checkCudaErrors(hipDeviceSynchronize());
    //float res = sum_wmma(input_h, input_size);
    float res_device = sum_device<80>(input_h, input_size);
    

    float res_cpu = (half)0.0;
    for(int i=0;i<input_size;i++)
        res_cpu+=__half2float(input_h[i]);
    std::cout<<"<-----------------------computing result----------------------->"<<std::endl;
    //std::cout<<"result of reduction with tensor core: "<<res<<std::endl;
    std::cout<<"result of reduction with whole device: "<<res_device<<std::endl;
    //std::cout<<"result of reduction with CUB: "<<res_cub<<std::endl;
    std::cout<<"result of reduction with CPU: "<<res_cpu<<std::endl;
    std::cout<<std::endl<<"<-----------------------all complete----------------------->"<<std::endl;

    free(input_h);
    free(input_h_cub);
    return 0;
}

