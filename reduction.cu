#include "hip/hip_runtime.h"
#include <assert.h>
#include <stdio.h>
#include <sstream>
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <mma.h>
#include <hip/hip_runtime_api.h>
#include <hipcub/hipcub.hpp> 

#define WMMA_M 16
#define WMMA_N 16
#define WMMA_K 16

#define SHMEM_SIZE 32*1024
#define WARP_SIZE 32
#define WARPS_PER_BLOCK 8
#define THREADS_PER_BLOCK (WARP_SIZE * WARPS_PER_BLOCK)

#define CONST_BYTES 16*16*2
#define SKEW_HALF 8                      //offset for avoding bank conflict
#define SHMEM_STRIDE 16+SKEW_HALF
#define INPUT_STORE_POINT WMMA_M
#define FREE_USE INPUT_STORE_POINT+16

#define checkCudaErrors(status) {                                      \
    std::stringstream _error;                                          \
    if (status != 0) {                                                 \
      _error << "Cuda failure\nError: " << hipGetErrorString(status); \
      std::stringstream _where, _message;                                \
      _where << __FILE__ << ':' << __LINE__;                             \
      _message << _error.str() + "\n" << __FILE__ << ':' << __LINE__;\
      std::cerr << _message.str() << "\nAborting...\n";                  \
      hipDeviceReset();                                                 \
      exit(EXIT_FAILURE);                                                \
    }                                                                  \
}

#define checkKernelErrors(expr)                             \
  do {                                                      \
    expr;                                                   \
                                                            \
    hipError_t __err = hipGetLastError();                 \
    if (__err != hipSuccess) {                             \
      printf("Line %d: '%s' failed: %s\n", __LINE__, #expr, \
             hipGetErrorString(__err));                    \
      abort();                                              \
    }                                                       \
  } while (0)



using namespace nvcuda;

// malloc gpu constant memory
__constant__  half P_d[16*16];

half P_h[16*16]={1.0,1.0,1.0,1.0,1.0,1.0,1.0,1.0,1.0,1.0,1.0,1.0,1.0,1.0,1.0,1.0,
                0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,
                0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,
                0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,
                0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,
                0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,
                0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,
                0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,
                0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,
                0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,
                0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,
                0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,
                0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,
                0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,
                0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,
                0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0
                };



__global__ void compute_reductions16N_warp(const half *input, float *output, int N){

    extern __shared__ half shmem[][16 + SKEW_HALF];
    half *free_use = (half*)&shmem[FREE_USE][0];

    const unsigned int warpId = threadIdx.x / WARP_SIZE;
    const unsigned int laneId = threadIdx.x % WARP_SIZE;

    if(warpId==0){

      typedef int4 copy_t;//vector pointer for fast copy
      //load P matrix to shared memory
      int shmem_row = laneId/2;
      //just remember this fixed copy approach!!
      copy_t *lane_ptr = (copy_t *)(P_d+laneId*8);          //one thread copy a int4 = 16bytes = 8 fp16.
      *((copy_t *)&shmem[shmem_row][0]+laneId%2) = *lane_ptr;
      
      //load input
      if(laneId < N<<1){
        lane_ptr = (copy_t *)(input+laneId*8);
        *((copy_t *)&shmem[INPUT_STORE_POINT+shmem_row][0]+laneId%2) = *lane_ptr;
      }
      else{
        *((copy_t *)&shmem[INPUT_STORE_POINT+shmem_row][0]+laneId%2) = make_int4(0,0,0,0);//padding with 0;
      }

       __syncthreads();  


      wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, half, wmma::row_major> P_frag;
      wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, half, wmma::col_major> PT_frag;
      wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, half, wmma::col_major> A_frag;
      wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, half, wmma::row_major> V_frag;
      wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, float> C_frag;
      wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, half> Vn_frag;
      wmma::fill_fragment(C_frag, 0.0f);

      
      wmma::load_matrix_sync(P_frag, &shmem[0][0], SHMEM_STRIDE);
      wmma::load_matrix_sync(PT_frag, &shmem[0][0], SHMEM_STRIDE);
      //wmma::load_matrix_sync(A_frag, &shmem[INPUT_STORE_POINT][0], SHMEM_STRIDE);
      wmma::load_matrix_sync(A_frag, &shmem[INPUT_STORE_POINT][0], SHMEM_STRIDE);

      wmma::mma_sync(Vn_frag, P_frag, A_frag, C_frag);//perform V = P x A

      wmma::store_matrix_sync(free_use, Vn_frag, 16, wmma::mem_row_major);//store Vn to shared memory
      wmma::load_matrix_sync(V_frag, free_use, 16);//load V from Vn

      wmma::mma_sync(C_frag, V_frag, PT_frag, C_frag);//perform output = V x PT 
      
      wmma::store_matrix_sync(output, C_frag, 16, wmma::mem_row_major);
    }

    __syncthreads();
    if(threadIdx.x==0)
      printf("kernel complete!\n");
}

/************************
 * WARP-LEVEL REDUCTION *
 ************************/
__global__ void compute_reductions256N_warp(const half *input, float *output, int N){

  extern __shared__ half shmem[][16 + SKEW_HALF];
  half *free_use = (half*)&shmem[FREE_USE][0];
  const unsigned int warpId = threadIdx.x / WARP_SIZE;
  const unsigned int laneId = threadIdx.x % WARP_SIZE;


  if(warpId==0){

    typedef int4 copy_t;//vector pointer for fast copy
    //load P matrix to shared memory
    int shmem_row = laneId/2;
    //just remember this fixed copy approach!!
    copy_t *lane_ptr = (copy_t *)(P_d+laneId*8);          //one thread copy a int4 = 16bytes = 8 fp16.
    *((copy_t *)&shmem[shmem_row][0]+laneId%2) = *lane_ptr;

     __syncthreads();  

    wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, half, wmma::row_major> P_frag;
    wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, half, wmma::col_major> PT_frag;
    wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, half, wmma::col_major> A_frag;
    wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, half, wmma::row_major> V_frag;
    wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, float> C_frag;
    wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, half> Vn_frag;
    wmma::fill_fragment(C_frag, 0.0f);
    //wmma::fill_fragment(Vn_frag, 0.0f);
    
    wmma::load_matrix_sync(P_frag, &shmem[0][0], SHMEM_STRIDE);
    wmma::load_matrix_sync(PT_frag, &shmem[0][0], SHMEM_STRIDE);
    //load input to frags
    for(int i=0;i<N;i++){
      wmma::load_matrix_sync(A_frag, input+i*256, 16);  
      wmma::mma_sync(Vn_frag, P_frag, A_frag, Vn_frag);//perform Vn = P x An+Vn-1
    }

    wmma::store_matrix_sync(free_use, Vn_frag, 16, wmma::mem_row_major);//store Vn to shared memory, because as an accumulator frag Vn cannot be used for computing multiplication
    wmma::load_matrix_sync(V_frag, free_use, 16);//load V from Vn as a matrix_a type 
  
    wmma::mma_sync(C_frag, V_frag, PT_frag, C_frag);//perform output = V x PT 
    
    wmma::store_matrix_sync(output, C_frag, 16, wmma::mem_row_major);

  }

  
  //if(threadIdx.x==0)
    //printf("%f kernel complete!\n", (float)input[N*256-1]);
}


/*************************
 * BLOCK-LEVEL REDUCTION *
 *************************/
__global__ void compute_reductions256N_block(const half *input, float *output, N){
  extern __shared__ half shmem[][16 + SKEW_HALF];
  half *free_use = (half*)&shmem[FREE_USE][0];
  const unsigned int warpId = threadIdx.x / WARP_SIZE;
  const unsigned int laneId = threadIdx.x % WARP_SIZE;

  if(warpId==0){
    typedef int4 copy_t;//vector pointer for fast copy
    //load P matrix to shared memory
    int shmem_row = laneId/2;
    //just remember this fixed copy approach!!
    copy_t *lane_ptr = (copy_t *)(P_d+laneId*8);          //one thread copy a int4 = 16bytes = 8 fp16.
    *((copy_t *)&shmem[shmem_row][0]+laneId%2) = *lane_ptr;

     __syncthreads();
  }

  wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, half, wmma::row_major> P_frag;
  wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, half, wmma::col_major> PT_frag;
  wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, half, wmma::col_major> A_frag;
  wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, half, wmma::row_major> V_frag;
  wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, float> C_frag;
  wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, half> Vn_frag;
  wmma::fill_fragment(C_frag, 0.0f);
  //wmma::fill_fragment(Vn_frag, 0.0f);
  
  wmma::load_matrix_sync(P_frag, &shmem[0][0], SHMEM_STRIDE);
  wmma::load_matrix_sync(PT_frag, &shmem[0][0], SHMEM_STRIDE);
  unsigned int i=0;
  while(warpId+i){
    wmma::load_matrix_sync(A_frag, input+i*256, 16);
    wmma::mma_sync(Vn_frag, P_frag, A_frag, Vn_frag);//perform Vn = P x An+Vn-1
    i+=WARPS_PER_BLOCK;
  }

}




//test the memory transfer speed
__global__ void shared_to_frag(const half *input, float *output, const int N){
  extern __shared__ half shmem[][16 + SKEW_HALF];
  const unsigned int warpId = threadIdx.x / WARP_SIZE;
  const unsigned int laneId = threadIdx.x % WARP_SIZE;

  if(warpId==0){
    typedef int4 copy_t;//vector pointer for fast copy
    //load P matrix to shared memory
    int shmem_row = laneId/2;
    copy_t *lane_ptr = (copy_t *)(P_d+laneId*8);       
    *((copy_t *)&shmem[shmem_row][0]+laneId%2) = *lane_ptr;
    
    __syncthreads();  
    //load input
    for(int i=0;i<100;i++){
      if(laneId < N<<1){
        copy_t *lane_ptr = (copy_t *)(input+laneId*8);
        *((copy_t *)&shmem[INPUT_STORE_POINT+shmem_row][0]+laneId%2) = *lane_ptr;
      }
      else{
        *((copy_t *)&shmem[INPUT_STORE_POINT+shmem_row][0]+laneId%2) = make_int4(0,0,0,0);//padding with 0;
      }
    }
     __syncthreads();  
     
    //wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, half, wmma::col_major> A_frag[32];

    //for(int i=0;i<1000;i++)
    //  wmma::load_matrix_sync(A_frag[0], &shmem[INPUT_STORE_POINT][0], 16);
    
  } 
  
}
/*
事实证明global->shared->frag和global->frag速度没什么却别
计算一次mma的时间约为load一个frag或从constant读到shared的10倍
*/





__host__ void init_input(half *input, int size){
  for(float i=0.0;i<size;i++){
    *(input+(int)i) = (half)(1.10);
  }
}


int main(){

    half *input_h;
    int N = (2<<10);
    int input_size = N*256;

    float *output_h;
    half *input_d;
    float *output_d;

    output_h = (float*)malloc(2*CONST_BYTES);
    input_h = (half*)malloc(2*input_size);
    init_input(input_h, input_size);

    //malloc GPU and copy contant data to constant memory
    checkCudaErrors(hipMalloc(&input_d, 2*input_size));
    checkCudaErrors(hipMalloc(&output_d, CONST_BYTES*2));
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(P_d), P_h, CONST_BYTES));

    //copy input to gpu
    checkCudaErrors(hipMemcpy(input_d, input_h, 2*input_size, hipMemcpyHostToDevice));

    //launch kernel
    //for(int i=0;i<100;i++)
    //checkKernelErrors( (compute_reductions16N_warp<<<1, THREADS_PER_BLOCK, SHMEM_SIZE>>>(input_d, output_d, 15)) );
    checkKernelErrors( (compute_reductions256N_warp<<<1, THREADS_PER_BLOCK, SHMEM_SIZE>>>(input_d, output_d, N)) );
    checkCudaErrors(hipDeviceSynchronize());

    //copy result back to cpu
    checkCudaErrors(hipMemcpy(output_h, output_d, 2*CONST_BYTES, hipMemcpyDeviceToHost) );

    
    //check the computing result
    for(int i=0;i<16;++i){
        for(int j=0;j<16;++j){
            std::cout<<output_h[16*i+j]<<",";
        }
        std::cout<<std::endl;
    }
   
    std::cout<<std::endl<<"all complete!"<<(float)input_h[N*256-1]<<std::endl;

    //free host&device memory
    free(output_h);
    checkCudaErrors(hipFree(input_d));
    hipFree((output_d));

    return 0;
}

//遗留问题 256N超过一个256就不对
//block-level reduction